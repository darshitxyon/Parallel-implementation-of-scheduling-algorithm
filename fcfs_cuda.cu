#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
 
__global__ void fcfs(int *b, int * w, int * t)
{
	int k;
	int j=threadIdx.x;
    extern __shared__ int XY[];
    if(i< blockDim-1 ){
        XY[threadIdx.x + 1 ] =b[j]; }

    if(threadIdx.x==0) XY[0]= 0.0; 

    for(unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
        __syncthreads();
        XY[threadIdx.x]+= XY[threadIdx.x - stride]; }

    w[j]=XY[threadIdx.x];
        
	   /*for(k=0;k<j;k++)
    	   {
             w[j]=w[j]+b[k];
            }
        */
	__syncthreads();
	t[j]=b[j]+w[j];
}


int main()
{
    
    int bt[20],p[20],wt[20],tat[20],i,j,n,total=0,total1=0,pos,temp;
    float avg_wt,avg_tat;
    printf("Enter number of process:");
    scanf("%d",&n);
 
    //ENTER THE VALUES
    printf("\nEnter Burst Time:\n");
    for(i=0;i<n;i++)
    {
        printf("p%d:",i+1);
        scanf("%d",&bt[i]);
        p[i]=i+1;           //contains process number
    }
    int *d_bt,*d_wt,*d_tat;
    clock_t begin = clock();

    hipMalloc( (void**)&d_bt, n * sizeof(int) ) ;
    hipMalloc( (void**)&d_wt, n * sizeof(int) ) ;
    hipMalloc( (void**)&d_tat, n * sizeof(int) ) ;

    hipMemcpy( d_bt, bt, n * sizeof(int), hipMemcpyHostToDevice ) ;

    fcfs<<<1,n>>>(d_bt,d_wt,d_tat);

    hipMemcpy( wt,d_wt, n * sizeof(int), hipMemcpyDeviceToHost ) ;
    hipMemcpy( tat,d_tat, n * sizeof(int), hipMemcpyDeviceToHost ) ;
    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    for(i=0;i<n;i++){
        total+=wt[i];
        total1+=tat[i];
        }
    avg_wt=(float)total/n;
    avg_tat=(float)total1/n;
    printf("\nProcess\t    Burst Time    \tWaiting Time\tTurnaround Time");
    for(i=0;i<n;i++)
    {
        printf("\np[%d]\t\t  %d\t\t    %d\t\t\t%d",i+1,bt[i],wt[i],tat[i]);
    }
    printf("\n\nAverage Waiting Time=%f",avg_wt);
    printf("\nAverage Turnaround Time=%f\n",avg_tat);
    printf("\ntime elapsed in execution=%f\n",time_spent);
    hipFree( d_bt );
    hipFree( d_wt );
    hipFree( d_tat );
	return 0;
}






