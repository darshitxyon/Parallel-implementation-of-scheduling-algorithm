#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
 
__global__ void sjf(int *b, int * pr, int * w, int * t)

{
int i=2*threadIdx.x;
int j=threadIdx.x;
int temp;
int pos=0;
if(i<(blockDim.x/2))
{
    if(b[i]<b[i+1]){ 
	temp=b[i];
	b[i]=b[i+1];
	b[i+1]=temp;	
	
	temp=pr[i];
	pr[i]=pr[i+1];
	pr[i+1]=temp;
	
    }
    if(b[i+1]<b[i+2]){ 
	temp=b[i+1];
	b[i+1]=b[i+2];
	b[i+2]=temp;
	
	temp=pr[i+1];
	pr[i+1]=pr[i+2];
	pr[i+2]=temp;

    }
}
__syncthreads();
/*w[0]=0;
for(k=0;k<j;k++)
    {w[j]+=b[k];}*/
extern __shared__ int XY[];
    if(i< blockDim-1 ){
        XY[threadIdx.x + 1 ] =b[j]; }

    if(threadIdx.x==0) XY[0]= 0.0; 

    for(unsigned int stride = 1; stride <= threadIdx.x; stride *= 2) {
        __syncthreads();
        XY[threadIdx.x]+= XY[threadIdx.x - stride]; }

    w[j]=XY[threadIdx.x];
    __syncthreads();
    t[j]=b[j]+w[j];

}


int main()
{
    
    int bt[20],p[20],wt[20],tat[20],i,j,n,total=0,total1=0,pos,temp;
    float avg_wt,avg_tat;
    printf("Enter number of process:");
    scanf("%d",&n);
 
    //ENTER THE VALUES
    printf("\nEnter Burst Time:\n");
    for(i=0;i<n;i++)
    {
        printf("p%d:",i+1);
        scanf("%d",&bt[i]);
        p[i]=i+1;           //contains process number
    }
    int *d_bt,*d_p,*d_wt,*d_tat;
    clock_t begin = clock();

     hipMalloc( (void**)&d_bt, n * sizeof(int) ) ;
     hipMalloc( (void**)&d_p, n * sizeof(int) ) ;
     hipMalloc( (void**)&d_wt, n * sizeof(int) ) ;
     hipMalloc( (void**)&d_tat, n * sizeof(int) ) ;

     hipMemcpy( d_bt, bt, n * sizeof(int), hipMemcpyHostToDevice ) ;
     hipMemcpy( d_p, p, n * sizeof(int), hipMemcpyHostToDevice ) ;

    sjf<<<1,n>>>(d_bt,d_p,d_wt,d_tat);

    hipMemcpy( bt,d_bt, n * sizeof(int), hipMemcpyDeviceToHost ) ;
    hipMemcpy( p,d_p, n * sizeof(int), hipMemcpyDeviceToHost ) ;
     hipMemcpy( wt,d_wt, n * sizeof(int), hipMemcpyDeviceToHost ) ;
    hipMemcpy( tat,d_tat, n * sizeof(int), hipMemcpyDeviceToHost ) ;
    
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    for(i=0;i<n;i++){
        total+=wt[i];
        total1+=tat[i];
        }
    avg_wt=(float)total/n;
    avg_tat=(float)total1/n;
    printf("\nProcess\t    Burst Time    \tWaiting Time\tTurnaround Time");
    for(i=0;i<n;i++)
    {
        printf("\np%d\t\t  %d\t\t    %d\t\t\t%d",p[i],bt[i],wt[i],tat[i]);
    }
    printf("\n\nAverage Waiting Time=%f",avg_wt);
    printf("\nAverage Turnaround Time=%f\n",avg_tat);
    printf("\ntime elapsed in execution=%f\n",time_spent);
	return 0;
}





    

